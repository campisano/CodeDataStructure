#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#define VECTOR_SIZE 10000000
#define THREADSperBLOCK 1024

// Kernel: usando mais de um bloco (uso t�pico)
__global__ void vecAdd(const float *A, const float *B, float *C)
{
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < VECTOR_SIZE)
		C[i] = A[i] + B[i];
}

int main(int argc, char** argv)
{
	unsigned int i;
	unsigned int size = VECTOR_SIZE * sizeof(float);
	float *h_A, *h_B, *h_C, *h_Cref;
	float *d_A, *d_B, *d_C;
	srand((unsigned int)time(NULL));
	double startTime, elapsedTimeCPU, elapsedTimeGPU;

	// Allocate CPU
	h_A = (float*) malloc(size);
	h_B = (float*) malloc(size);
	h_C = (float*) malloc(size);
	h_Cref = (float*) malloc(size);

	// Initialize CPU
	for(i = 0; i < VECTOR_SIZE; i++){
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Compute CPU
	startTime = clock();
	for(i = 0; i < VECTOR_SIZE; i++)
		h_Cref[i] = h_A[i] + h_B[i];

	elapsedTimeCPU = (clock() - startTime) / CLOCKS_PER_SEC;
	printf("CPU = %f s\n",elapsedTimeCPU);

	// Allocate GPU
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	// Copy to GPU
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Compute GPU
	startTime = clock();
	vecAdd<<< (VECTOR_SIZE+THREADSperBLOCK-1)/THREADSperBLOCK, THREADSperBLOCK >>>(d_A, d_B, d_C);
	hipDeviceSynchronize(); // Aguarda fim da execu��o do kernel pela GPU
	elapsedTimeGPU = (clock() - startTime) / CLOCKS_PER_SEC;
	printf("GPU = %f s\n", elapsedTimeGPU);

	printf("Speedup = %f X\n", (elapsedTimeCPU / elapsedTimeGPU));

	// CPU waits for kernel's run end (GPU)
	// Copy results from GPU
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify result
	for (i = 0; i < VECTOR_SIZE; ++i)
		if (fabs(h_C[i] - h_Cref[i]) > 1e-5)
			break;

	printf("%s\n", (i == VECTOR_SIZE) ? "PASSED!" : "FAILED!");

	// free memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_Cref);

	// pause
	getch();

	return 0;
}