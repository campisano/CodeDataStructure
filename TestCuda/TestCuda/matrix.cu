#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#define N 10000 // Matrix dim = N x N
#define THREADSperBLOCK 1024

// Kernel: usando mais de um bloco (uso t�pico)
__global__ void vecAdd(const float *A, const float *B, float *C)
{
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

	int i = N * y + x;

	if(i < N * N)
	{
		C[i] = A[i] + B[i];
	}
}

int main(int argc, char** argv)
{
	// vars
	unsigned int VECTOR_SIZE = N * N;
	unsigned int size = VECTOR_SIZE * sizeof(float);
	float *h_A, *h_B, *h_C, *h_Cref;
	float *d_A, *d_B, *d_C;
	
	double startTime, elapsedTimeCPU, elapsedTimeGPU;

	// init random
	srand((unsigned int)time(NULL));

	// Allocate CPU
	{
		h_A = (float*) malloc(size);
		h_B = (float*) malloc(size);
		h_C = (float*) malloc(size);
		h_Cref = (float*) malloc(size);
	}

	// Initialize CPU
	{
		for(int i = 0; i < VECTOR_SIZE; ++i){
			h_A[i] = rand() / (float)RAND_MAX;
			h_B[i] = rand() / (float)RAND_MAX;
		}
	}

	// Compute CPU
	{
		startTime = clock();

		for (int i = 0; i < (VECTOR_SIZE); ++i)
		{
			h_Cref[i] = h_A[i] + h_B[i];
		}

		elapsedTimeCPU = (clock() - startTime) / CLOCKS_PER_SEC;
		printf("CPU = %f s\n",elapsedTimeCPU);
	}

	// Allocate GPU
	{
		hipMalloc((void**)&d_A, size);
		hipMalloc((void**)&d_B, size);
		hipMalloc((void**)&d_C, size);
	}

	// Copy to GPU
	{
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	}

	// Compute GPU
	{
		startTime = clock();
		vecAdd<<< (VECTOR_SIZE+THREADSperBLOCK-1)/THREADSperBLOCK, THREADSperBLOCK >>>(d_A, d_B, d_C);
		hipDeviceSynchronize(); // Aguarda fim da execu��o do kernel pela GPU
		elapsedTimeGPU = (clock() - startTime) / CLOCKS_PER_SEC;
		printf("GPU = %f s\n", elapsedTimeGPU);
	}

	printf("Speedup = %f X\n", (elapsedTimeCPU / elapsedTimeGPU));

	// CPU waits for kernel's run end (GPU)
	// Copy results from GPU
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify result
	{
		int i;
		for (i = 0; i < VECTOR_SIZE; ++i)
			if (fabs(h_C[i] - h_Cref[i]) > 1e-5)
				break;

		printf("%s\n", (i == VECTOR_SIZE) ? "PASSED!" : "FAILED!");
	}

	// free memory
	{
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		free(h_A);
		free(h_B);
		free(h_C);
		free(h_Cref);
	}

	// pause
	getch();

	return 0;
}