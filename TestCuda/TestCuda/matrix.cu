#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#define THREADSperBLOCK 1024

__global__ void vecMult(const float *A, const float *B, float *C, unsigned int width)
{
	//unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	/*
	int i = N * y + x;

	if(i < N * N)
	{
		C[i] = A[i] + B[i];
	}
	*/

	// Kernel: usando SOMENTE UM BLOCO que possui 1024 threads
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	float sum = 0, a, b;
	
	for (int k = 0; k < width; ++k)
	{
		a = A[ty * width + k]; // a = h_A[y * width + k];
		b = B[k * width + tx]; // b = h_B[k * width + x];
		sum += a*b;
	}

	C[ty * width + tx] = sum; //h_Cref[y * width + x] = sum;
}

int main(int argc, char** argv)
{
	// vars
	unsigned int width = 32;
	unsigned long VECTOR_SIZE = width * width;
	unsigned long size = VECTOR_SIZE * sizeof(float);
	float *h_A, *h_B, *h_C, *h_Cref;
	float *d_A, *d_B, *d_C;
	
	double startTime, elapsedTimeCPU, elapsedTimeGPU;

	// init random
	srand((unsigned int)time(NULL));

	// Allocate CPU
	{
		h_A = (float*) malloc(size);
		h_B = (float*) malloc(size);
		h_C = (float*) malloc(size);
		h_Cref = (float*) malloc(size);
	}

	// Initialize CPU
	{
		for(int i = 0; i < VECTOR_SIZE; ++i){
			h_A[i] = rand() / (float)RAND_MAX;
			h_B[i] = rand() / (float)RAND_MAX;
		}
	}

	// Compute CPU
	{
		startTime = clock();
		float sum, a, b;
		for (int y = 0; y < width; ++y)
		{
			for (int x = 0; x < width; ++x)
			{
				sum = 0;

				for (int k = 0; k < width; ++k)
				{
					a = h_A[y * width + k];
					b = h_B[k * width + x];
					sum += a*b;
				}

				h_Cref[y * width + x] = sum;
			}
		}

		elapsedTimeCPU = (clock() - startTime) / CLOCKS_PER_SEC;
		printf("CPU = %f s\n",elapsedTimeCPU);
	}

	// Allocate GPU
	{
		hipMalloc((void**)&d_A, size);
		hipMalloc((void**)&d_B, size);
		hipMalloc((void**)&d_C, size);
	}

	// Copy to GPU
	{
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	}

	// Compute GPU
	{
		dim3 dimBlock(width, width);
		dim3 dimGrid(1, 1);
		startTime = clock();
		vecMult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);
		hipDeviceSynchronize(); // Aguarda fim da execu��o do kernel pela GPU
		elapsedTimeGPU = (clock() - startTime) / CLOCKS_PER_SEC;
		printf("GPU = %f s\n", elapsedTimeGPU);
	}

	printf("Speedup = %f X\n", (elapsedTimeCPU / elapsedTimeGPU));

	// CPU waits for kernel's run end (GPU)
	// Copy results from GPU
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify result
	{
		int i;
		for (i = 0; i < VECTOR_SIZE; ++i)
			if (fabs(h_C[i] - h_Cref[i]) > 1e-5)
				break;

		printf("%s\n", (i == VECTOR_SIZE) ? "PASSED!" : "FAILED!");
	}

	// free memory
	{
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		free(h_A);
		free(h_B);
		free(h_C);
		free(h_Cref);
	}

	// pause
	getch();

	return 0;
}